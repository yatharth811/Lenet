#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <vector>
#include <string>
#include <stdlib.h>
#include <cassert>
#include <chrono>
#include "utils.hpp"

using namespace std::chrono;

typedef float float1D;
typedef float float2D;
typedef float float3D;
typedef float float4D;

// Weights and Biases
float3D conv1_wts[500];
float1D conv1_bias[20];
float4D conv2_wts[25000];
float1D conv2_bias[50];
float4D fclayer1_wts[400000];
float1D fclayer1_bias[500];
float4D fclayer2_wts[5000];
float1D fclayer2_bias[10];

// Number of Streams
#define NUM_STREAMS 32
hipStream_t stream[NUM_STREAMS];

// Gpu constant memory for weights
__constant__ float2D kernel_2D[500];

__global__ void conv2DKernel_shared(float2D *input, float2D *conv, int n, int f, float bias, int filter_num){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Shared Memory Optimization
  __shared__ float N_ds[28][28];
  if(row<n && col<n){
    N_ds[row][col] = input[row*n+col];
  }
  __syncthreads();

  if (row < n - f + 1 and col < n - f + 1) {
    float temp = bias;
    for (int k = 0; k < f; k += 1) {
      for (int l = 0; l < f; l += 1) {
        assert(filter_num*25+k*f+l<500);
        temp += kernel_2D[filter_num*25+k*f+l] * N_ds[k+row][l+col];
      }
    }
    conv[row * (n - f + 1) + col] = temp;
  }
  
}

__global__ void conv3DKernel_shared(float3D *input, float3D *kernel, float2D *conv, int n, int f, int channels, float bias) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  // Shared Memory Optimization
  __shared__ float N_ds[12][12][20];

  if(row<n && col<n){
    for (int channel = 0; channel < channels; channel += 1) {
      N_ds[row][col][channel] = input[channel * n * n +  row * n + col];
    }
  }
  __syncthreads();

  if (row < n - f + 1 and col < n - f + 1) {
    float temp = 0;
    for (int k = 0; k < f; k += 1) {
      for (int l = 0; l < f; l += 1) {
        for (int channel = 0; channel < channels; channel += 1) {
          temp += kernel[channel * f * f + k * f + l]*N_ds[k+row][l+col][channel];
        }
      }
    }
    conv[row * (n - f + 1) + col] = temp + bias;
  }
}

__global__ void max_pool_kernel(float *input, float *output, int n, int f){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int out_height = n/f;
  int out_width = n/f;
  if(row<out_height && col<out_width){
    float temp = -1e9;
    int st_row = row*f, st_col = col*f;
    for(int i=0; i<f; i++){
        for(int j=0; j<f; j++){
            temp = max(temp,input[(st_row+i)*n+(st_col+j)]);
        }
    }
    output[row*out_width+col] = temp;
  }
}

__global__ void fclayer1_kernel(float3D *input, float3D *output, float4D* weights, float *bias){
  int row = threadIdx.x;

  // Shared Memory Optimization
  __shared__ float N_ds[800];

  if(row<800){
    N_ds[row] = input[row];
  }

  __syncthreads();

  if(row<500){
    float temp = bias[row];
    for (int j = 0; j < 50; j += 1) {
      for (int k = 0; k < 4; k += 1) {
        for (int l = 0; l < 4; l += 1) {
          temp += (N_ds[j * 16 + k * 4 + l] * weights[row * 800 + j * 16 + k * 4 + l]);
        }
      }
    }
    output[row] = (temp > 0 ? temp : 0);
  }
}

__global__ void fclayer2_kernel(float3D *input, float3D *output, float4D* weights, float *bias){
  int row = threadIdx.x;
  // Shared Memory Optimization
  __shared__ float N_ds[500];

  if(row<500){
    N_ds[row] = input[row];
  }

  __syncthreads();

  if(row<10){
    float temp = bias[row];
    for (int j = 0; j < 500; j += 1) {
      temp += (N_ds[j] * weights[row * 500 + j]);
    }
    output[row] = temp;
  }
}

void conv2D(float2D *input, float2D *conv, int n, int f, float bias, int filter_num, int stream_id) {

  // Dim
  dim3 block_dim(32, 32);
  
  conv2DKernel_shared<<<1, block_dim, 0, stream[stream_id]>>>(input, conv, n, f, bias, filter_num);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
}



void conv3D(float3D *input, float3D* kernel, float2D *conv, int n, int f, int channels, float bias, int stream_id) {

  // Dim
  dim3 block_dim(32, 32);

  // Call the kernel
  conv3DKernel_shared<<<1, block_dim, 0, stream[stream_id]>>>(input, kernel, conv, n, f, channels, bias);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));

}


// LeNet 5 Architecture
// 28 * 28 image input (*) 20 5x5 filters + 20 * bias
// input: float[28][28]
// filters: float[20][5][5]
// output: float[20][24][24]
void conv1(float2D *input, float3D *conv, float1D* bias, int stream_id) {
  for (int i = 0; i < 20; i += 1) {
    conv2D(input, conv + i * 24 * 24, 28, 5, bias[i], i, stream_id);
  }
}

// Max pooling here
// 20 input 24 x 24 and 2x2 kernel -> 20 output 12x12
// input: float[20][24][24]
// output: float[20][12][12]
void pool1(float3D *input, float3D *output, int stream_id) {
  dim3 block_dim(32, 32);
  for (int i = 0; i < 20; i += 1) {
    max_pool_kernel<<<1,block_dim,0,stream[stream_id]>>>(input + i * 24 * 24, output + i * 12 * 12, 24, 2);
  }
}


// 2nd convolutional layer
// input: float[20][12][12]
// filters: float[50][20][5][5]
// output: float[50][8][8]
void conv2(float3D *input, float3D *output, float4D *weights, float *bias, int stream_id) {
  for (int i = 0; i < 50; i += 1) {
    conv3D(input, weights+i*500, output+i*64, 12, 5, 20, bias[i], stream_id);
  }
}

// Max pooling here
// input: float[50][8][8]
// output: float[50][4][4]
void pool2(float3D *input, float3D *output, int stream_id) {
  dim3 block_dim(32, 32);
  for (int i = 0; i < 50; i += 1) {
    max_pool_kernel<<<1, block_dim, 0, stream[stream_id]>>>(input + i * 64, output + i * 16, 8, 2);
  }
}

// FCLayer 1 + ReLu
// input: float[50][4][4]
// filters: float[500][50][4][4]
// output: float[500][1][1]
void fclayer1(float3D *input, float3D *output, float4D* weights, float *bias, int stream_id) {
  fclayer1_kernel<<<1, 1024, 0, stream[stream_id]>>>(input, output, weights, bias);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));

}

// FCLayer 2 + ReLu
// input: float[500][1][1]
// filters: float[10][500][1][1]
// output: float[10][1][1]
void fclayer2(float3D *input, float3D *output, float4D *weights, float *bias, int stream_id) {
  fclayer2_kernel<<<1, 512, 0, stream[stream_id]>>>(input, output, weights, bias);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
}

// Output layer
// input: float[10][1][1]
int outputLayer(float3D* input) {
  int classLabel{-1};
  float prob = 0.0F;
  for (int i = 0; i < 10; i += 1) {
    if (input[i] > prob) {
      prob = input[i];
      classLabel = i;
    }
  }
  return classLabel;
}

int main(int argc, char** argv) {

  if(argc!=2){
    std::cerr << "Wrong Arguments Passed, pass 0 (stream off) or 1 (stream on)\n";
    return 0;
  }

  int is_stream = std::stoi(argv[1]);
  // Images file name
  std::string filename = "img_path.txt";
  auto start = high_resolution_clock::now();

  // load weights and biases.

  if (!load_weights(conv1_wts, conv1_bias, 500, 20, "weights/conv1.txt")) {
    std::cout << "Unable to load conv1 weights." << std::endl;
    return 0;
  }
  hipMemcpyToSymbol(HIP_SYMBOL(kernel_2D), conv1_wts, 500 * sizeof(float),0,hipMemcpyHostToDevice);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error1: %s\n", hipGetErrorString(err));
  
  if (!load_weights(conv2_wts, conv2_bias, 25000, 50, "weights/conv2.txt")) {
    std::cout << "Unable to load conv2 weights." << std::endl;
    return 0;
  }
  
  if (!load_weights(fclayer1_wts, fclayer1_bias, 400000, 500, "weights/fc1.txt")) {
    std::cout << "Unable to load fclayer1 weights." << std::endl;
    return 0;
  }
  
  if (!load_weights(fclayer2_wts, fclayer2_bias, 5000, 10, "weights/fc2.txt")) {
    std::cout << "Unable to load fclayer2 weights." << std::endl;
    return 0;
  }

  // Allocate Pinned Memory
  float2D *image;
  hipHostMalloc((void**)&image, 784*NUM_STREAMS*sizeof(float));

  float3D *fclayer2_output;
  hipHostMalloc((void**)&fclayer2_output, 10*NUM_STREAMS*sizeof(float));
 
  // Allocate Device Memory
  float3D *d_conv1_output;
  hipMalloc(&d_conv1_output,11520*NUM_STREAMS*sizeof(float));

  float3D *d_pool1_output;
  hipMalloc(&d_pool1_output,2880*NUM_STREAMS*sizeof(float));

  float4D *d_conv2_wts;
  hipMalloc(&d_conv2_wts,25000*sizeof(float));
  hipMemcpy(d_conv2_wts,conv2_wts,25000*sizeof(float),hipMemcpyHostToDevice);

  float3D *d_conv2_output;
  hipMalloc(&d_conv2_output,3200*NUM_STREAMS*sizeof(float));

  float3D *d_pool2_output;
  hipMalloc(&d_pool2_output,800*NUM_STREAMS*sizeof(float));

  float3D *d_fclayer1_output;
  hipMalloc(&d_fclayer1_output,500*NUM_STREAMS*sizeof(float));

  float3D *d_fclayer1_wts;
  hipMalloc(&d_fclayer1_wts,400000*sizeof(float));
  hipMemcpy(d_fclayer1_wts,fclayer1_wts,400000*sizeof(float),hipMemcpyHostToDevice);

  float3D *d_fclayer1_bias;
  hipMalloc(&d_fclayer1_bias,500*sizeof(float));
  hipMemcpy(d_fclayer1_bias,fclayer1_bias,500*sizeof(float),hipMemcpyHostToDevice);

  float3D *d_fclayer2_output;
  hipMalloc(&d_fclayer2_output,10*NUM_STREAMS*sizeof(float));

  float3D *d_fclayer2_wts;
  hipMalloc(&d_fclayer2_wts,5000*sizeof(float));
  hipMemcpy(d_fclayer2_wts,fclayer2_wts,5000*sizeof(float),hipMemcpyHostToDevice);

  float3D *d_fclayer2_bias;
  hipMalloc(&d_fclayer2_bias,10*sizeof(float));
  hipMemcpy(d_fclayer2_bias,fclayer2_bias,10*sizeof(float),hipMemcpyHostToDevice);

  float2D *d_image;
  hipMalloc(&d_image,784*NUM_STREAMS*sizeof(float));

  // Load image names and their labels
  std::vector<std::string> images = load_image_paths(filename);
  std::vector<std::vector<float>> images_prob;

  // Initialize cuda streams
  for (int i = 0; i < NUM_STREAMS; ++i)
    hipStreamCreate(&stream[i]);

  if(is_stream){

    int num_itr = (images.size()+NUM_STREAMS-1)/NUM_STREAMS;
    for(int i=0; i<num_itr; i++){
      int st = i*NUM_STREAMS, en = std::min((i+1)*NUM_STREAMS,(int)images.size());
      // Distributing images to streams
      for(int j=st; j<en; j++){
          int idx = j%NUM_STREAMS;
          if (!load_image(image+idx*784, 784,"pre-proc-img/"+images[j])) {
              std::cout << "Unable to load image " << "pre-proc-img/"+images[j] << std::endl;
              return 0;
          }
          // CNN Architecture
          hipMemcpyAsync(d_image+idx*784,image+idx*784,784*sizeof(float),hipMemcpyHostToDevice,stream[idx]);
          conv1(d_image+idx*784, d_conv1_output+idx*11520, conv1_bias, idx);
          pool1(d_conv1_output+idx*11520, d_pool1_output+idx*2880, idx);
          conv2(d_pool1_output+idx*2880, d_conv2_output+idx*3200, d_conv2_wts, conv2_bias,idx);
          pool2(d_conv2_output+idx*3200, d_pool2_output+idx*800, idx);
          fclayer1(d_pool2_output+idx*800, d_fclayer1_output+idx*500, d_fclayer1_wts, d_fclayer1_bias, idx);
          fclayer2(d_fclayer1_output+idx*500, d_fclayer2_output+idx*10, d_fclayer2_wts, d_fclayer2_bias, idx);
          hipMemcpyAsync(fclayer2_output+idx*10, d_fclayer2_output+idx*10, 10*sizeof(float), hipMemcpyDeviceToHost, stream[idx]);
      }
      // Wait for computation to finish in all streams
      hipDeviceSynchronize();
      // Generate prediction
      for(int j=st; j<en; j++){
        int idx = j%NUM_STREAMS;
        std::vector<float> prob = top5_prob(fclayer2_output+idx*10,10);
        images_prob.push_back(prob);
      }
    }
  }
  else{
    
    for(int i=0; i<images.size(); i++){
      if (!load_image(image, 784,"pre-proc-img/"+images[i])) {
          std::cout << "Unable to load image " << "pre-proc-img/"+images[i] << std::endl;
          return 0;
      }
      // CNN Architecture
      hipMemcpyAsync(d_image,image,784*sizeof(float),hipMemcpyHostToDevice,stream[0]);
      conv1(d_image, d_conv1_output, conv1_bias, 0);
      pool1(d_conv1_output, d_pool1_output, 0);
      conv2(d_pool1_output, d_conv2_output, d_conv2_wts, conv2_bias, 0);
      pool2(d_conv2_output, d_pool2_output, 0);
      fclayer1(d_pool2_output, d_fclayer1_output, d_fclayer1_wts, d_fclayer1_bias, 0);
      fclayer2(d_fclayer1_output, d_fclayer2_output, d_fclayer2_wts, d_fclayer2_bias, 0);
      hipMemcpyAsync(fclayer2_output, d_fclayer2_output, 10*sizeof(float), hipMemcpyDeviceToHost, stream[0]);
      hipDeviceSynchronize();
      std::vector<float> prob = top5_prob(fclayer2_output,10);
      images_prob.push_back(prob);
    }
  }

  for(int i=0; i<images.size(); i++) writeToFile(images_prob[i],"output/"+images[i]+".txt");

  auto stop = high_resolution_clock::now();
  auto duration = duration_cast<milliseconds>(stop - start);
  std::cout << "Time taken to process 10000 images in ms = " << duration.count() << "\n";
  
  // Free pinned memory and device memory 
  hipHostFree(image);
  hipHostFree(fclayer2_output);
  hipFree(d_image);
  hipFree(d_conv1_output);
  hipFree(d_conv2_output);
  hipFree(d_pool1_output);
  hipFree(d_pool2_output);
  hipFree(d_fclayer1_output);
  hipFree(d_fclayer1_wts);
  hipFree(d_fclayer1_bias);
  hipFree(d_fclayer2_output);
  hipFree(d_fclayer2_wts);
  hipFree(d_fclayer2_bias);

  return 0;
}